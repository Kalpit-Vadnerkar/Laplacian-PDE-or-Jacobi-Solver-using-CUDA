#include "hip/hip_runtime.h"
#include "jacobian_kernel.h"

#define BLOCK 32

__global__ 
void laplacePDE(float *d_in, float *d_temp, int numRows, int numCols, float *d_error){

 /*
   Your kernel here: Make sure to check for boundary conditions
  */

  int x = threadIdx.x + blockIdx.x * blockDim.x;
  int y = threadIdx.y + blockIdx.y * blockDim.y;

  int Offset = y * numCols + x;

  for (int i = 0; i < 100; ++i){
    if (x < numCols - 1 && x > 0 && y < numRows - 1 && y > 0){
        d_temp[Offset] = (d_in[(y - 1) * numCols + x] + d_in[y * numCols + x - 1] + d_in[y * numCols + x + 1] + d_in[(y + 1) * numCols + x]) / 4;
    }
    __syncthreads();
    if (x < numCols && y < numRows){
        d_error[0] += abs(d_temp[Offset] - d_in[Offset]);
        __syncthreads();
        d_in[Offset] = d_temp[Offset];
    }
    if (d_error[0] < 0.000001f)
        break;
  }
}




void launch_jacobian(float* d_in, float* d_temp, const int numRows, const int numCols, float* d_error){
    // configure launch params here 
    
    dim3 block(BLOCK, BLOCK, 1);
    dim3 grid((numCols-1)/BLOCK + 1, (numRows-1)/BLOCK + 1, 1);

    laplacePDE<<<grid,block>>>(d_in, d_temp, numRows, numCols, d_error);
    hipDeviceSynchronize();
    checkCudaErrors(hipGetLastError());
}